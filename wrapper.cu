#include<cuda_error_check.h>
#include<hip/hip_runtime.h>

#include <stdlib.h>
#include <stdio.h>

#define NDIM (3)
void allocate_device_memory( double ** d_pos, double ** d_acc, double ** d_vel, double ** d_mass , int n ) {

  double * temp = NULL;
  cuda_error_check( hipMalloc( (void **)&temp, NDIM*n*sizeof(double) ));
  *d_pos = temp;

  temp = NULL;
  cuda_error_check( hipMalloc( (void **)&temp, NDIM*n*sizeof(double) ));
  *d_vel = temp;

  temp = NULL;
  cuda_error_check( hipMalloc( (void **)&temp, NDIM*n*sizeof(double) ));
  *d_acc = temp;

  temp = NULL;
  cuda_error_check( hipMalloc( (void **)&temp, n*sizeof(double) ));
  *d_mass = temp;
}

void free_device_memory(double ** d_pos, double ** d_acc, double ** d_vel, double ** d_mass)  {
  hipFree(*d_pos);
  *d_pos = NULL;
  hipFree(*d_vel);
  *d_vel = NULL;
  hipFree(*d_acc);
  *d_acc = NULL;
  hipFree(*d_mass);
  *d_mass = NULL;
  cuda_error_check ( hipGetLastError() );
}

void transfer_to_device(double* dst, double* src, int n) {
  cuda_error_check( hipMemcpy(dst, src, n*sizeof(double), hipMemcpyHostToDevice) );
}
